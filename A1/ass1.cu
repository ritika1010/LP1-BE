#include "hip/hip_runtime.h"
#include<iostream>
#include<math.h>

#define n 8

using namespace std;

__global__ void minimum(int *input){
  
  int tid = threadIdx.x;
  int step_size = 1;
  int number_of_threads = blockDim.x;

  cout<<"Number of threads needed are "<<number_of_threads;

  while(number_of_threads > 0){
    if(tid < number_of_threads){
      int first = tid * step_size * 2;
      int second = first + step_size;
      if(input[second] < input[first]){
        input[first] = input[second];
      }
    }
    step_size <<= 1;
    number_of_threads >>= 1;
  }
}

void randintgen(int *arr, int size){
  for(int i=0;i<size;i++){
    arr[i] = rand()%100;
    cout<<arr[i]<<" ";
  }
  cout<<endl;
}

int main(){

  int sizearr = n * sizeof(int);

  int *arr, *arr_d, result;

  arr = (int *)malloc(sizearr);

  randintgen(arr,n);

  hipMalloc((void **)&arr_d,sizearr);

  hipMemcpy(arr_d,arr,sizearr,hipMemcpyHostToDevice);

  minimum<<1,n/2>>(arr_d);

  hipMemcpy(&result,arr_d,sizeof(int),hipMemcpyDeviceToHost);

  cout<<"Minimum is "<<result<<endl;
  return 0;
}
